//Pierre Chouteau & Elisa Delhommé

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



//Partie 1 - Prise en main de Cuda (03 décembre 2021)

//Création d'une matrice (p lignes, n colonnes)
void MatrixInit(float *M, int n, int p){
    
    float random_value;
    
    //Valeurs entre -1 et 1
    for(int i=0; i<n*p; i++){
        random_value = (float)rand()/(float)(RAND_MAX/1.0);
        M[i] =  random_value;
    }
}


//Affichage d'une matrice
void MatrixPrint(float *M, int n, int p){
    
    for(int i=0; i<p; i++){
        for(int j=i*n; j<n*(i+1); j++){
            printf("%f ", M[j]);
        }
        printf("\n");
    }
}


//Addition de deux matrices sur CPU
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
    for(int i=0; i<n*p;i++){
        Mout[i] = M1[i]+M2[i];
    }
    
}

//Addition de deux matrices sur GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
}

//Multiplication de deux matrices NxN sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n){
    
}


//Multiplication de deux matrices NxN sur GPU
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
    
}

//Fonction main
int main(){
    
    printf("Hello from the CPU!\n\n");
    
    //Test de MatrixInit et MatrixPrint
    printf("Here is our matrix!\n\n");
    
    float *M;    
    
    int n = 3;
    int p = 3;
    
    //Allocation de la mémoire pour la création de la matrice
    M = (float*)malloc(n * p * sizeof(float));
    
    MatrixInit(M, n, p);
    MatrixPrint(M, n, p);
    
    printf("\n");
    
    
    //Test de MatrixAdd
    float *M1;
    float *M2;
    float *Mout;
    
    //Allocation des mémoires
    M1 = (float*)malloc(n * p * sizeof(float));
    M2 = (float*)malloc(n * p * sizeof(float));
    Mout = (float*)malloc(n * p * sizeof(float));
    
    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);
    MatrixAdd(M1, M2, Mout, n, p);
    
    printf("Matrice 1\n");
    MatrixPrint(M1, n, p);
    printf("\nMatrice 2\n");
    MatrixPrint(M2, n, p);
    printf("\nMatrice résultante de la somme:\n");
    MatrixPrint(Mout, n, p);
}