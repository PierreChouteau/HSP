//Pierre Chouteau & Elisa Delhommé

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



//Partie 1 - Prise en main de Cuda (03 décembre 2021)

//Création d'une matrice (n lignes, p colonnes)
void MatrixInit(float *M, int n, int p){
        
    float random_value;
    
    //Valeurs entre -1 et 1
    for (int i = 0; i < n * p; i++){
        random_value = (float)rand() / (float)(RAND_MAX/1.0);
        M[i] =  random_value;
    }
}


//Affichage d'une matrice
void MatrixPrint(float *M, int n, int p){
        
    for (int lig = 0 ; lig < n; lig++){
        for(int col = lig * p; col < p * (lig+1); col++){
            printf("%f ", M[col]);
        }
        printf("\n");
    }
}


//Addition de deux matrices sur CPU
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
    printf("Addition from the CPU...\n\n");
    
    for (int i = 0; i < n * p; i++){
        Mout[i] = M1[i] + M2[i];
    }
    
}

//Addition de deux matrices sur GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
    printf("Addition from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (lig < n && col < p){
        Mout[lig * p + col] = M1[lig * p + col] + M2[lig * p + col];
    }
}

//Multiplication de deux matrices NxN sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n){
    
    printf("Multiplication from the CPU...\n\n");
    
    for (int lig = 0; lig < n; lig++){
        for (int col = 0; col < n; col++){
            float s = 0.0f;
            for (int i = 0; i < n; i++) {
                s += M1[lig * n + i] * M2[i * n + col];
            }
            Mout[lig * n + col] = s;
        }
    }
}


//Multiplication de deux matrices NxN sur GPU
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
    printf("Multiplication from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float s = 0.0f;
    
    if (lig < n && col < n){
        for (int i = 0; i < n; i++){
            s += M1[lig * n + i] * M2[i * n + col];
        }
        Mout[lig * n + col] = s;
    }
}


//Multiplication d'une matrice NxP avec une PxM sur GPU
__global__ void cudaMatrixMultGeneral(float *M1, float *M2, float *Mout, int n, int p, int m){
    printf("Multiplication from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float s = 0.0f;
    
    if (lig < n && col < m){
        for (int i = 0; i < p; i++){
            s += M1[lig * p + i] * M2[i * m + col];
        }
        Mout[lig * m + col] = s;
    }
}

//Fonction main
int main(){
    
    /////////////// TOUT CE QUI SE PASSE ICI EST FAIT DU CPU \\\\\\\\\\\\\\\
    
    //Test de MatrixInit et MatrixPrint
    
    float *M;
    
    int n = 3;
    int p = 2;
    int m = 3;
    
    //Allocation de la mémoire pour la création de la matrice
    M = (float*)malloc(n * p * sizeof(float));
    
    MatrixInit(M, n, p);
    //MatrixPrint(M, n, p);
    
    //printf("\n");
    
    free(M);
    
    //Test de MatrixAdd
    float *M1;
    float *M2;
    float *Mout;
    
    //Allocation des mémoires
    M1 = (float*)malloc(n * p * sizeof(float));
    M2 = (float*)malloc(p * m * sizeof(float));
    Mout = (float*)malloc(n * m * sizeof(float));
    
    MatrixInit(M1, n, p);
    MatrixInit(M2, p, m);
    //MatrixAdd(M1, M2, Mout, n, p);
    //MatrixMult(M1, M2, Mout, n);
    
//     printf("Matrice 1\n");
//     MatrixPrint(M1, n, p);
//     printf("\nMatrice 2\n");
//     MatrixPrint(M2, n, p);
//     printf("\nMatrice résultante de la mutliplication:\n");
//     MatrixPrint(Mout, n, p);

    
    
    
    /////////////// TOUT CE QUI SE PASSE ICI EST FAIT DU GPU \\\\\\\\\\\\\\\
    
    //Test de cudaMatrixAdd
    float *d_M1, *d_M2, *d_Mout;
    
    //Allocation des mémoires des matrices pour cuda
    hipMalloc((void**)&d_M1, sizeof(float) * n * p);
    hipMalloc((void**)&d_M2, sizeof(float) * p * m);
    hipMalloc((void**)&d_Mout, sizeof(float) * n * m);

    hipMemcpy(d_M1, M1, sizeof(float) * n * p, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, sizeof(float) * p * m, hipMemcpyHostToDevice);

    //Addition sur GPU
    dim3 block_size(n, m);
    dim3 grid_size(1, 1);
    // cudaMatrixAdd<<<grid_size, block_size>>>(d_M1, d_M2, d_Mout, n, p);
    
    //Multiplication sur GPU    
    cudaMatrixMultGeneral<<<grid_size,block_size>>>(d_M1, d_M2, d_Mout, n, p, m);
    hipDeviceSynchronize();
    
    
    //Copie du résultat sur CPU
    hipMemcpy(Mout, d_Mout, sizeof(float) * n * m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    printf("Matrice 1\n");
    MatrixPrint(M1, n, p);
    printf("\nMatrice 2\n");
    MatrixPrint(M2, p, m);
    printf("\nMatrice résultante de la Multiplication:\n");
    MatrixPrint(Mout, n, m);
    
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);
    
    free(M1);
    free(M2);
    free(Mout);
}