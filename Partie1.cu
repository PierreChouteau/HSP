// Pierre Chouteau & Elisa Delhommé


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



// Partie 1 - Prise en main de Cuda (03 décembre 2021)

/*
*** Function Name : MatrixInit ***

Sert à initialiser n'importe quelle matrice de taille NxP avec des valeurs entre -1 et 1 

Paramètres : 
    n : nombre de lignes de la matrice,
    p : nombre de colonnes de la matrice si n différent de p,
    M : pointeur de la matrice
*/
void MatrixInit(float *M, int n, int p){
        
    float random_value;
    
    // Valeurs entre -1 et 1
    for (int i = 0; i < n * p; i++){
        random_value = (float)rand() / (float)(RAND_MAX/1.0);
        M[i] =  random_value;
    }
}


/*
*** Function Name : MatrixPrint ***

Sert à afficher une matrice NxP dans une forme plus conventionnelle. 

                                                              0 0 0
ex : M = [0 0 0; 0 0 0; 0 0 0] sera affichée comme suit : M = 0 0 0   
                                                              0 0 0 

Paramètres : 
    n : nombre de lignes de la matrice,
    p : nombre de colonnes de la matrice si n différent de p,
    M : pointeur de la matrice
*/
void MatrixPrint(float *M, int n, int p){
        
    for (int lig = 0 ; lig < n; lig++){
        for(int col = lig * p; col < p * (lig+1); col++){
            printf("%f ", M[col]);
        }
        printf("\n");
    }
}


/*
*** Function Name : MatrixAdd ***

Sert à additionner deux matrices de même taille NxP sur le CPU

Paramètres : 
    n : nombre de lignes des matrice,
    p : nombre de colonnes des matrice si n différent de p,
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille NxP,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxP
*/
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
    printf("Addition from the CPU...\n\n");
    
    for (int i = 0; i < n * p; i++){
        Mout[i] = M1[i] + M2[i];
    }
    
}

/*
*** Function Name : cudaMatrixAdd ***

Sert à additionner deux matrices de même taille NxP sur le GPU 

Paramètres : 
    n : nombre de lignes des matrice,
    p : nombre de colonnes des matrices si n différent de p,
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille NxP,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxP,
    
On peut considérer les dimensions des matrices comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks,
    les colonnes correspondent aux threads
*/
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    
    printf("Addition from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (lig < n && col < p){
        Mout[lig * p + col] = M1[lig * p + col] + M2[lig * p + col];
    }
}

/*
*** Function Name : MatrixMult***

Sert à effectuer la multiplication matricielle (dot) de deux matrices carrées NxN sur CPU

Paramètres : 
    n : nombre de lignes et colonne des matrices,
    M1 : pointeur de la matrice 1 de taille NxN,
    M2 : pointeur de la matrice 2 de taille NxN,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxN,
*/
void MatrixMult(float *M1, float *M2, float *Mout, int n){
    
    printf("Multiplication from the CPU...\n\n");
    
    for (int lig = 0; lig < n; lig++){
        for (int col = 0; col < n; col++){
            float s = 0.0f;
            for (int i = 0; i < n; i++) {
                s += M1[lig * n + i] * M2[i * n + col];
            }
            Mout[lig * n + col] = s;
        }
    }
}


/*
*** Function Name : cudaMatrixMult ***

Sert à effectuer la multiplication matricielle (dot) de deux matrices carrées NxN sur GPU

Paramètres : 
    n : nombre de lignes et de colonnes des matrices,
    M1 : pointeur de la matrice 1 de taille NxN,
    M2 : pointeur de la matrice 2 de taille NxN,
    Mout : pointeur vers la matrice résultante de la multiplication de taille NxN,
    
On peut considérer les dimensions des matrices comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks,
    les colonnes correspondent aux threads
*/
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
    printf("Multiplication from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float s = 0.0f;
    
    if (lig < n && col < n){
        for (int i = 0; i < n; i++){
            s += M1[lig * n + i] * M2[i * n + col];
        }
        Mout[lig * n + col] = s;
    }
}


/*
*** Function Name : cudaMatrixMultGeneral ***

Sert à effectuer la multiplication matricielle (dot) d'une matrice NxP avec une matrice PxM sur le GPU

Paramètres : 
    n : nombre de lignes de la matrice M1
    p : nombre de colonnes de M1, de lignes de M2
    m : nombre de colonnes de M2
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille PxM,
    Mout : pointeur vers la matrice résultante de la multiplication de taille NxM

On peut considérer les dimensions de la matrice de sortie comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks : n
    les colonnes correspondent aux threads : m
*/
__global__ void cudaMatrixMultGeneral(float *M1, float *M2, float *Mout, int n, int p, int m){
    printf("Multiplication from the GPU...\n\n");
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float s = 0.0f;
    
    if (lig < n && col < m){
        for (int i = 0; i < p; i++){
            s += M1[lig * p + i] * M2[i * m + col];
        }
        Mout[lig * m + col] = s;
    }
}

// Fonction main
int main(){
    
    /////////////// TOUT CE QUI SE PASSE ICI EST FAIT DU CPU \\\\\\\\\\\\\\\
    //**********************************************************************
    
    // Test de MatrixInit et MatrixPrint
    float *M;
    
    int n = 3;
    int p = 2;
    int m = 3;
    
    // Allocation de la mémoire pour la création de la matrice
    M = (float*)malloc(n * p * sizeof(float));
    
    MatrixInit(M, n, p);
//    MatrixPrint(M, n, p);
//    printf("\n");
    
    free(M);
    
    // Test de MatrixAdd
    float *M1;
    float *M2;
    float *Mout;
    
    // Allocation des mémoires
    M1 = (float*)malloc(n * p * sizeof(float));
    M2 = (float*)malloc(p * m * sizeof(float));
    Mout = (float*)malloc(n * m * sizeof(float));
    
    MatrixInit(M1, n, p);
    MatrixInit(M2, p, m);
    
    // Test de MatrixAdd et MatrixMult sur CPU
//    MatrixAdd(M1, M2, Mout, n, p);
//    MatrixMult(M1, M2, Mout, n);
    
//     printf("Matrice 1\n");
//     MatrixPrint(M1, n, p);
//     printf("\nMatrice 2\n");
//     MatrixPrint(M2, n, p);
//     printf("\nMatrice résultante de la mutliplication:\n");
//     MatrixPrint(Mout, n, p);

    
    
    // Test de cudaMatrixAdd
    float *d_M1, *d_M2, *d_Mout;
    
    // Allocation des mémoires des matrices pour cuda
    hipMalloc((void**)&d_M1, sizeof(float) * n * p);
    hipMalloc((void**)&d_M2, sizeof(float) * p * m);
    hipMalloc((void**)&d_Mout, sizeof(float) * n * m);

    hipMemcpy(d_M1, M1, sizeof(float) * n * p, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, sizeof(float) * p * m, hipMemcpyHostToDevice);
    
    
    /////////////// TOUT CE QUI SE PASSE ICI EST FAIT DU GPU \\\\\\\\\\\\\\\
    //**********************************************************************
    
    dim3 block_size(n, m);
    dim3 grid_size(1, 1);
    
    // Addition sur GPU
//    cudaMatrixAdd<<<grid_size, block_size>>>(d_M1, d_M2, d_Mout, n, p);
    
    // Multiplication sur GPU    
    cudaMatrixMultGeneral<<<grid_size,block_size>>>(d_M1, d_M2, d_Mout, n, p, m);
    hipDeviceSynchronize();
    
    
    // Copie du résultat sur CPU
    hipMemcpy(Mout, d_Mout, sizeof(float) * n * m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // Affichage du résultat de la multiplication
    printf("Matrice 1\n");
    MatrixPrint(M1, n, p);
    printf("\nMatrice 2\n");
    MatrixPrint(M2, p, m);
    printf("\nMatrice résultante de la Multiplication:\n");
    MatrixPrint(Mout, n, m);
    
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);
    
    free(M1);
    free(M2);
    free(Mout);
}