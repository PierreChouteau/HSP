#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

int main(int argc, char *argv[]){

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    int block_size = atoi(argv[2]);
    int grid_size = atoi(argv[1]);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("Grid size : %d\n", grid_size);
    printf("Block size : %d\n", block_size);
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}
